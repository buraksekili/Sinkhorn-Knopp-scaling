
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <omp.h>
#include <math.h>       /* fabsf */
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define DEBUG 0

//Error check-----
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
//Error check-----
//This is a very good idea to wrap your calls with that function.. Otherwise you will not be able to see what is the error.
//Moreover, you may also want to look at how to use cuda-memcheck and cuda-gdb for debugging.

__global__ void calculateError(int* xadj, int* adj, double* rv, double* cv, double* maxErr, int maxOperation) {
	// Get idx for each thread
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < maxOperation) {
		int starti = xadj[i], endi = xadj[i+1];
		double err = 0;
		for (int j = starti; j < endi; j++) err += rv[i] * cv[adj[j]];

		err = fabs(1-err);
		if (err > *maxErr) *maxErr = err;
	} 
}

__global__ void scaleskRV(int* xadj, int* adj, double* rv, double* cv, int maxOperation) {
	// Get idx for each thread
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < maxOperation) {
		int starti = xadj[i], endi = xadj[i+1];
		double rowSum = 0;
		for (int j = starti; j < endi; j++) rowSum += cv[adj[j]];
		rv[i] = 1 / rowSum;	
	}
}


__global__ void scaleskCV(int* txadj, int* tadj, double* rv, double* cv, int maxOperation) {
	// Get idx for each thread
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < maxOperation) {
		int	starti = txadj[i], endi = txadj[i+1];
		double colSum = 0;
		for (int j = starti; j < endi; j++) colSum += rv[tadj[j]];
		cv[i] = 1 / colSum;
	} 
}

void wrapper(int* adj, int* xadj, int* tadj, int* txadj, double* rv, double* cv, int* nov, int* nnz, int siter){
	// Initialize rv and cv
	for (int i = 0; i < *nov; i++) rv[i] = cv[i] = 1;

	// Transfer data from host to device.
	int NO_THREADS = 1024;
	int NO_BLOCKS = (*nov + NO_THREADS - 1)/NO_THREADS;

	int maxOperation = (*nov) - 1;
	std::cout << "NO_BLOCKS " << NO_BLOCKS << std::endl;
	std::cout << "NO_THREADS " << NO_THREADS << std::endl;
	std::cout << "NO_THREADS * NO_BLOCKS " << NO_THREADS * NO_BLOCKS << std::endl;
	std::cout << "maxOperation " << maxOperation << std::endl;
	std::cout << "no of one " << xadj[*nov] << std::endl;

	int* adj_d, *xadj_d, *tadj_d, *txadj_d;
	gpuErrchk(hipMalloc( (void**) &adj_d, (*nnz) * sizeof(int)));
	gpuErrchk(hipMemcpy(adj_d, adj, (*nnz) * sizeof(int), hipMemcpyHostToDevice ));

	gpuErrchk(hipMalloc( (void**) &xadj_d, (*nov) * sizeof(int)));
	gpuErrchk(hipMemcpy(xadj_d, xadj, (*nov) * sizeof(int), hipMemcpyHostToDevice ));

	gpuErrchk(hipMalloc( (void**) &tadj_d, (*nnz) * sizeof(int)));
	gpuErrchk(hipMemcpy(tadj_d, tadj,(*nnz) * sizeof(int), hipMemcpyHostToDevice ));

	gpuErrchk(hipMalloc( (void**) &txadj_d, (*nov) * sizeof(int)));
	gpuErrchk(hipMemcpy(txadj_d, txadj,(*nov) * sizeof(int), hipMemcpyHostToDevice ));

	double* rv_d, *cv_d;
	gpuErrchk(hipMalloc( (void**) &rv_d, (*nov) * sizeof(double)));
	gpuErrchk(hipMemcpy(rv_d, rv, (*nov) * sizeof(double), hipMemcpyHostToDevice ));

	gpuErrchk(hipMalloc( (void**) &cv_d, (*nov) * sizeof(double)));
	gpuErrchk(hipMemcpy(cv_d, cv, (*nov) * sizeof(double), hipMemcpyHostToDevice ));

	double* err_d;
	double* err = new double(0);

	hipEvent_t start, stop;
  	hipEventCreate(&start);
  	hipEventRecord(start, 0);

	for (int i = 0; i < siter; i++) {
		// Fill rv 
		scaleskRV<<<NO_BLOCKS, NO_THREADS>>>(xadj_d, adj_d, rv_d, cv_d, maxOperation);
		gpuErrchk(hipPeekAtLastError());

		// Fill cv 
		scaleskCV<<<NO_BLOCKS, NO_THREADS>>>(txadj_d, tadj_d, rv_d, cv_d, maxOperation);
		gpuErrchk(hipPeekAtLastError());

		// calculate error
		gpuErrchk(hipMalloc((void**) &err_d, sizeof(double)));
		calculateError<<<NO_BLOCKS, NO_THREADS>>>(xadj_d, adj_d, rv_d, cv_d, err_d, maxOperation);

		// get error from device
		gpuErrchk(hipMemcpy(err, err_d, sizeof(double), hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(err_d));

		std::cout << "iter " << i << " - error: " << *err << std::endl;
	}
  
  	hipEventCreate(&stop);
  	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop);
  
	float elapsedTime;
 	hipEventElapsedTime(&elapsedTime, start, stop);
 	printf("GPU scale took: %f s\n", elapsedTime/1000);
    
	gpuErrchk(hipFree(xadj_d));
	gpuErrchk(hipFree(adj_d));
	gpuErrchk(hipFree(txadj_d));
	gpuErrchk(hipFree(tadj_d));
	gpuErrchk(hipFree(rv_d));
	gpuErrchk(hipFree(cv_d));
  
}

